#include "hip/hip_runtime.h"
// ref: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

#include <stdint.h>
#include <cstddef>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_helper.hpp"
#include "reduction_gpu.hpp"
#include "reduction.hpp"


using namespace std;



__global__ void reduce_gpu_v4_kernel(const uint32_t *g_data, uint32_t*g_out){
    extern __shared__ uint32_t sdata[];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    sdata[tid] = g_data[i] + g_data[i+blockDim.x];

    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) {
        g_out[blockIdx.x] = sdata[0];
    } 
}

/*
uint32_t reduce_gpu_v4(const uint32_t *d_data, uint32_t *h_tmp_out, uint32_t *d_tmp_out, size_t n, int n_blocks, int n_threads, hipDeviceProp_t &prop) {
    size_t n_blocks_v4 = n_blocks/2;
    reduce_gpu_v4_kernel<<<n_blocks_v4, n_threads, sizeof(uint32_t)*n_threads>>>(d_data, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(uint32_t)*n_blocks_v4, hipMemcpyDefault));
    uint32_t ret = reduce(h_tmp_out, n_blocks_v4);
    return ret;
}

*/