#include "hip/hip_runtime.h"
// ref: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

#include <stdint.h>
#include <cstddef>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_helper.hpp"
#include "reduction_gpu.hpp"
#include "reduction.hpp"

using namespace std;

template <uint32_t kBlockSize>
__device__ void warpReduce(volatile int *sdata, int tid)
{
    assert(warpSize == 32);
    int v = sdata[tid];
    if (kBlockSize >= 64)
    {
        v += sdata[tid + 32];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
    if (kBlockSize >= 32)
    {
        v += sdata[tid + 16];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
    if (kBlockSize >= 16)
    {
        v += sdata[tid + 8];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
    if (kBlockSize >= 8)
    {
        v += sdata[tid + 4];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
    if (kBlockSize >= 4)
    {
        v += sdata[tid + 2];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
    if (kBlockSize >= 2)
    {
        v += sdata[tid + 1];
        __syncwarp();
        sdata[tid] = v;
        __syncwarp();
    }
}

__global__ void reduce_gpu_old_v1_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sdata[tid] = g_in[i];
    }
    else
    {
        sdata[tid] = 0;
    }

    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v1(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    reduce_gpu_old_v1_kernel<<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

__global__ void reduce_gpu_old_v2_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sdata[tid] = g_in[i];
    }
    else
    {
        sdata[tid] = 0;
    }

    __syncthreads();

    for (uint32_t s = 1; s < blockDim.x; s *= 2)
    {
        uint32_t index = 2 * s * tid;
        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v2(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    reduce_gpu_old_v2_kernel<<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

__global__ void reduce_gpu_old_v3_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sdata[tid] = g_in[i];
    }
    else
    {
        sdata[tid] = 0;
    }

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v3(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    reduce_gpu_old_v3_kernel<<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

__global__ void reduce_gpu_old_v4_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int sum_value = 0;
    if (i < n)
    {
        sum_value = g_in[i];
    }
    if ((i + blockDim.x) < n)
    {
        sum_value += g_in[i + blockDim.x];
    }
    sdata[tid] = sum_value;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v4(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    reduce_gpu_old_v4_kernel<<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

template <uint32_t kBlockSize>
__global__ void reduce_gpu_old_v5_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int sum_value = 0;
    if (i < n)
    {
        sum_value = g_in[i];
    }
    if ((i + blockDim.x) < n)
    {
        sum_value += g_in[i + blockDim.x];
    }
    sdata[tid] = sum_value;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > warpSize; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid < warpSize)
    {
        warpReduce<kBlockSize>(sdata, tid);
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v5(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    assert(512 == n_threads);
    reduce_gpu_old_v5_kernel<512><<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

template <uint32_t kBlockSize>
__global__ void reduce_gpu_old_v6_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int sum_value = 0;
    if (i < n)
    {
        sum_value = g_in[i];
    }
    if ((i + blockDim.x) < n)
    {
        sum_value += g_in[i + blockDim.x];
    }
    sdata[tid] = sum_value;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 512; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (kBlockSize >= 512)
    {
        if (tid < 256)
        {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (kBlockSize >= 256)
    {
        if (tid < 128)
        {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (kBlockSize >= 128)
    {
        if (tid < 64)
        {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    assert(warpSize == 32);
    if (tid < 32)
    {
        warpReduce<kBlockSize>(sdata, tid);
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v6(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    assert(512 == n_threads);
    reduce_gpu_old_v6_kernel<512><<<n_blocks, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks);
    return ret;
}

template <uint32_t kBlockSize>
__global__ void reduce_gpu_old_v7_kernel(const int *g_in, size_t n, int *g_out)
{
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int grid_size = 2 * blockDim.x * gridDim.x;
    int sum_value = 0;

    while (i < n)
    {
        sum_value += g_in[i];
        if ((i + kBlockSize) < n)
        {
            sum_value += g_in[i + kBlockSize];
        }
        i += grid_size;
    }

    sdata[tid] = sum_value;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 512; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (kBlockSize >= 512)
    {
        if (tid < 256)
        {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (kBlockSize >= 256)
    {
        if (tid < 128)
        {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (kBlockSize >= 128)
    {
        if (tid < 64)
        {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    assert(warpSize == 32);
    if (tid < 32)
    {
        warpReduce<kBlockSize>(sdata, tid);
    }
    if (tid == 0)
    {
        g_out[blockIdx.x] = sdata[0];
    }
}

int reduce_gpu_old_v7(const int *d_in, int *h_tmp_out, int *d_tmp_out, size_t n, int n_blocks, int n_threads)
{
    size_t n_blocks_v7 = 128;
    assert(512 == n_threads);
    reduce_gpu_old_v7_kernel<512><<<n_blocks_v7, n_threads, sizeof(int) * n_threads>>>(d_in, n, d_tmp_out);
    checkCudaErrors(hipMemcpy(h_tmp_out, d_tmp_out, sizeof(int) * n_blocks_v7, hipMemcpyDefault));
    int ret = reduce(h_tmp_out, n_blocks_v7);
    return ret;
}